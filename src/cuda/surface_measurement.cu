#include "hip/hip_runtime.h"
// Measures the surface, i.e. computes vertex and normal maps from a depth frame
// This is CUDA code; compile with nvcc
// Author: Christian Diller, git@christian-diller.de

#include "include/common.h"
#include <opencv2/opencv.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/cudawarping.hpp>
#include <opencv2/cudafilters.hpp>
#include <opencv2/cudaarithm.hpp>
#include <opencv2/ximgproc/edge_filter.hpp>

using cv::cuda::GpuMat;

namespace kinectfusion {
    namespace internal {
        namespace cuda {

            __global__
            void kernel_compute_vertex_map(const PtrStepSz<unsigned char> gpu_mask, 
                                           const PtrStepSz<float> depth_map, PtrStep<float3> vertex_map,
                                           const float depth_cutoff, const CameraParameters cam_params)
            {
                const int x = blockIdx.x * blockDim.x + threadIdx.x;
                const int y = blockIdx.y * blockDim.y + threadIdx.y;
                if (x >= depth_map.cols || y >= depth_map.rows)
                    return;

                unsigned char mask_value = gpu_mask.ptr(y)[x];
                float depth_value = depth_map.ptr(y)[x];
                if (mask_value == 0 || depth_value > depth_cutoff || depth_value <= 10.0f) {
                    depth_value = 0.f;
                } // outlier filter out
                Vec3fda vertex(
                        (x - cam_params.principal_x) * depth_value / cam_params.focal_x,
                        (y - cam_params.principal_y) * depth_value / cam_params.focal_y,
                        depth_value);
                vertex_map.ptr(y)[x] = make_float3(vertex.x(), vertex.y(), vertex.z());
            }

            __global__
            void kernel_compute_vertex_map(const PtrStepSz<float> depth_map, PtrStep<float3> vertex_map,
                                           const float depth_cutoff, const CameraParameters cam_params)
            {
                const int x = blockIdx.x * blockDim.x + threadIdx.x;
                const int y = blockIdx.y * blockDim.y + threadIdx.y;

                if (x >= depth_map.cols || y >= depth_map.rows)
                    return;

                float depth_value = depth_map.ptr(y)[x];
                if (depth_value > depth_cutoff) depth_value = 0.f; // Depth cutoff

                Vec3fda vertex(
                        (x - cam_params.principal_x) * depth_value / cam_params.focal_x,
                        (y - cam_params.principal_y) * depth_value / cam_params.focal_y,
                        depth_value);

                vertex_map.ptr(y)[x] = make_float3(vertex.x(), vertex.y(), vertex.z());
            }

            __global__
            void kernel_compute_normal_map(const PtrStepSz<float3> vertex_map, PtrStep<float3> normal_map, const CameraParameters cam_params)
            {
                const int x = blockIdx.x * blockDim.x + threadIdx.x;
                const int y = blockIdx.y * blockDim.y + threadIdx.y;

                // disable all the corner cases introduced by gpu allocation.
                if (x < 1 || x >= vertex_map.cols - 1 || y < 1 || y >= vertex_map.rows - 1)
                    return;

                Vec3fda ray_dir(
                    (x - cam_params.principal_x) / cam_params.focal_x, // x component
                    (y - cam_params.principal_y) / cam_params.focal_y, // y component
                    1.0f); // z component
                ray_dir.normalize();
                
                const Vec3fda left(&vertex_map.ptr(y)[x - 1].x);
                const Vec3fda right(&vertex_map.ptr(y)[x + 1].x);
                const Vec3fda upper(&vertex_map.ptr(y - 1)[x].x);
                const Vec3fda lower(&vertex_map.ptr(y + 1)[x].x);

                Vec3fda normal;

                if (left.z() == 0 || right.z() == 0 || upper.z() == 0 || lower.z() == 0)
                    normal = Vec3fda(0.f, 0.f, 0.f);
                else {
                    Vec3fda hor(left.x() - right.x(), left.y() - right.y(), left.z() - right.z());
                    Vec3fda ver(upper.x() - lower.x(), upper.y() - lower.y(), upper.z() - lower.z());

                    normal = hor.cross(ver);
                    normal.normalize();

                    // switch normal to point to the outer side of the mesh, with negative z value
                    // here normal vector has already been normalized. -> length == 1
                    if (normal.z() > 0)
                        normal *= -1; 
                        
                    ray_dir *= -1; // let direction of ray points back to the camera.
                    float dot_product = ray_dir.x() * normal.x() + ray_dir.y() * normal.y() + ray_dir.z() * normal.z(); // these two value are all normalized. thus dot_product == cos_value
                    // the angle between inversed ray_dir and normal value's range is [0, 180) on a spherical surface
                    // thus using cosine function makes sense. and for angle less than 60 degree is regarded to be useful.
                    // which means, the cosine value between [1, 1/2] is useful. -> the dot product >= 1/2 is useful for reconstruction
                    if (dot_product < 0.80f) { // 0.8660254037844386 -> sqrt(3) / 2
                        normal = Vec3fda(0.f, 0.f, 0.f);
                    }
                }

                normal_map.ptr(y)[x] = make_float3(normal.x(), normal.y(), normal.z());
            }

            __global__
            void vanilla_kernel_compute_normal_map(const PtrStepSz<float3> vertex_map, PtrStep<float3> normal_map)
            {
                // for now, this normal calculation is clear enough
                const int x = blockIdx.x * blockDim.x + threadIdx.x;
                const int y = blockIdx.y * blockDim.y + threadIdx.y;

                // disable all the corner cases introduced by gpu allocation.
                if (x < 1 || x >= vertex_map.cols - 1 || y < 1 || y >= vertex_map.rows - 1)
                    return;

                const Vec3fda left(&vertex_map.ptr(y)[x - 1].x);
                const Vec3fda right(&vertex_map.ptr(y)[x + 1].x);
                const Vec3fda upper(&vertex_map.ptr(y - 1)[x].x);
                const Vec3fda lower(&vertex_map.ptr(y + 1)[x].x);

                Vec3fda normal;

                if (left.z() == 0 || right.z() == 0 || upper.z() == 0 || lower.z() == 0)
                    normal = Vec3fda(0.f, 0.f, 0.f);
                else {
                    Vec3fda hor(left.x() - right.x(), left.y() - right.y(), left.z() - right.z());
                    Vec3fda ver(upper.x() - lower.x(), upper.y() - lower.y(), upper.z() - lower.z());

                    normal = hor.cross(ver);
                    normal.normalize();

                    if (normal.z() > 0)
                        normal *= -1;
                }

                normal_map.ptr(y)[x] = make_float3(normal.x(), normal.y(), normal.z());
            }

            void compute_vertex_map(const GpuMat& gpu_mask, const GpuMat& depth_map, GpuMat& vertex_map, const float depth_cutoff,
                                    const CameraParameters cam_params)
            {
                dim3 threads(32, 32);
                dim3 blocks((depth_map.cols + threads.x - 1) / threads.x, (depth_map.rows + threads.y - 1) / threads.y);

                kernel_compute_vertex_map<<<blocks, threads>>>(gpu_mask, depth_map, vertex_map, depth_cutoff, cam_params);

                hipDeviceSynchronize();
            }

            void compute_vertex_map(const GpuMat& depth_map, GpuMat& vertex_map, const float depth_cutoff,
                                    const CameraParameters cam_params)
            {
                dim3 threads(32, 32);
                dim3 blocks((depth_map.cols + threads.x - 1) / threads.x, (depth_map.rows + threads.y - 1) / threads.y);

                kernel_compute_vertex_map<<<blocks, threads>>>(depth_map, vertex_map, depth_cutoff, cam_params);

                hipDeviceSynchronize();
            }

            void compute_normal_map(const GpuMat& vertex_map, GpuMat& normal_map, const CameraParameters cam_params)
            {
                dim3 threads(32, 32);
                dim3 blocks((vertex_map.cols + threads.x - 1) / threads.x,
                            (vertex_map.rows + threads.y - 1) / threads.y);

                kernel_compute_normal_map<<<blocks, threads>>>(vertex_map, normal_map, cam_params);

                hipDeviceSynchronize();
            }

            // use sobel to do image gradient calculation. (to test the great changes in depth image)
            void computeGradient(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& grad, int kernel_size, cv::cuda::Stream& stream) {
                // create sobel filter
                auto sobel_x = cv::cuda::createSobelFilter(input.type(), CV_32F, 1, 0, kernel_size);
                auto sobel_y = cv::cuda::createSobelFilter(input.type(), CV_32F, 0, 1, kernel_size);

                // calculate the gradient by x axis & y axis
                cv::cuda::GpuMat grad_x, grad_y;
                sobel_x->apply(input, grad_x);
                sobel_y->apply(input, grad_y);

                // calculate the magnitude of the depth input. -> by fuse gradient of x axis and y axis
                cv::cuda::magnitude(grad_x, grad_y, grad);
                stream.waitForCompletion();
            }

            // CUDA utils: generate mask by gradient calculation. 
            void generateGradientMask(const cv::cuda::GpuMat& depth, cv::cuda::GpuMat& mask, float gradient_threshold, cv::cuda::Stream& stream) {
                // compute the gradient by fusing x-axis & y-axis gradient
                cv::cuda::GpuMat grad;
                computeGradient(depth, grad, 3, stream);

                // generate the mask
                cv::cuda::GpuMat thresholded;
                cv::cuda::threshold(grad, thresholded, gradient_threshold, 255.0, cv::THRESH_BINARY_INV);

                // convert generated mask to unsigned char -> CV_8UC1
                thresholded.convertTo(mask, CV_8U);
                stream.waitForCompletion();
            }

            // CUDA utils: generate consistency mask
            void generateConsistencyMask(const cv::cuda::GpuMat& depth, cv::cuda::GpuMat& mask, float depth_threshold, cv::cuda::Stream& stream) {
                // create sliding window filter
                auto box_filter = cv::cuda::createBoxFilter(depth.type(), depth.type(), cv::Size(3, 3));

                // compute the mean of neighboring pixels.
                cv::cuda::GpuMat mean_depth;
                box_filter->apply(depth, mean_depth);

                // compute the differences of neighboring depth value
                cv::cuda::GpuMat diff;
                cv::cuda::absdiff(depth, mean_depth, diff);

                // generate threshold of the depth image
                cv::cuda::GpuMat thresholded;
                cv::cuda::threshold(diff, thresholded, depth_threshold, 255.0, cv::THRESH_BINARY_INV);

                // convert the mask to CV_8UC1 mask
                thresholded.convertTo(mask, CV_8U);
                stream.waitForCompletion();
            }

            // CUDA utils: generate IR consistency mask
            void generateIRConsistencyMask(const cv::cuda::GpuMat& depth, const cv::cuda::GpuMat& ir, cv::cuda::GpuMat& mask, float gradient_threshold, cv::cuda::Stream& stream) {
                // calculate the gradient of depth and IR map, respectively.
                cv::cuda::GpuMat depth_grad, ir_grad;
                computeGradient(depth, depth_grad, 3, stream);
                computeGradient(ir, ir_grad, 3, stream);

                // compute the difference of these two gradients.
                cv::cuda::GpuMat diff;
                cv::cuda::absdiff(depth_grad, ir_grad, diff);

                // generate mask.
                cv::cuda::GpuMat thresholded;
                cv::cuda::threshold(diff, thresholded, gradient_threshold, 255.0, cv::THRESH_BINARY_INV);

                // convert to one channel.
                thresholded.convertTo(mask, CV_8U);
                stream.waitForCompletion();
            }

            // caller.
            void generateEdgeMask(cv::cuda::GpuMat& depth_map, 
                      cv::cuda::GpuMat& ir_image, 
                      cv::cuda::GpuMat& existing_mask,
                      float gradient_threshold, // 10-30
                      float consistency_threshold, // 10-50 mm
                      cv::cuda::Stream& stream) {
                // ensure the scale consistency
                CV_Assert(depth_map.size() == ir_image.size());
                CV_Assert(depth_map.type() == CV_32FC1);
                CV_Assert(ir_image.type() == CV_8UC1);
                CV_Assert(existing_mask.size() == depth_map.size());
                CV_Assert(existing_mask.type() == CV_8UC1);

                cv::cuda::GpuMat gradient_mask;
                generateGradientMask(depth_map, gradient_mask, gradient_threshold, stream);

                cv::cuda::GpuMat consistency_mask;
                generateConsistencyMask(depth_map, consistency_mask, consistency_threshold, stream);

                cv::cuda::GpuMat ir_consistency_mask;
                generateIRConsistencyMask(depth_map, ir_image, ir_consistency_mask, gradient_threshold, stream);

                // combine the masks
                cv::cuda::GpuMat combined_mask;
                cv::cuda::bitwise_and(gradient_mask, consistency_mask, combined_mask, cv::noArray(), stream);
                cv::cuda::bitwise_and(combined_mask, ir_consistency_mask, combined_mask, cv::noArray(), stream);
                // write back to existing_mask
                cv::cuda::bitwise_and(existing_mask, combined_mask, existing_mask, cv::noArray(), stream);
                stream.waitForCompletion();
            }
        }
    }
}